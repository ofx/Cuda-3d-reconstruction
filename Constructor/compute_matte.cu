#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda_types.hpp>

#include <hip/hip_runtime.h>

#include <iostream>

#include "cuda_common.cuh"

#include "Exception.h"

__global__
void compute_matte_kernel(const uint3 color, const uint treshold, const uint tolerance, const cv::cuda::PtrStepSz<uchar3> in, cv::cuda::PtrStepSz<uchar> out)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < in.cols && y < in.rows)
	{
		double i_255 = 1.0 / 255.0;
		
		float r = float(in(y, x).x), g = float(in(y, x).y), b = float(in(y, x).z);

		float d = ((r * i_255 - color.x * i_255) * (r * i_255 - color.x * i_255)) + ((g * i_255 - color.y * i_255) * (g * i_255 - color.y * i_255)) + ((b * i_255 - color.z * i_255) * (b * i_255 - color.z * i_255));
		
		// Sqrt hack
		unsigned int i = *(unsigned int*)&(d);
		i += 127 << 23;
		i >>= 1;
		float distance = *(float*)&i;

		int grey = 255 * distance / sqrt(3.0);
		if (grey <= treshold)
		{
			out(y, x) = 0;
		}
		else if (grey >= tolerance)
		{
			out(y, x) = 255;
		}
		else
		{
			grey = 255 * (grey - treshold) / (tolerance - treshold);

			out(y, x) = grey;
		}
	}
}

void compute_matte(const uint3 color, const uint treshold, const uint tolerance, const cv::cuda::PtrStepSz<uchar3> in, cv::cuda::PtrStepSz<uchar> out)
{
	dim3 blockSize(128, 8);
	dim3 gridSize = dim3(iDivUp(in.cols, blockSize.x), iDivUp(in.rows, blockSize.y));

	compute_matte_kernel<<<gridSize, blockSize>>>(color, treshold, tolerance, in, out);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		// WTF! We randomly encounter invalid argument errors here while the block and grid sizes are valid,
		// Nsight debugger reports some random (OpenCV-related?) errors
		/*char b[500];
		sprintf(b, "Failed to compute matte: %s", hipGetErrorString(err));
		throw_line(b);*/
	}
}