#include <hip/hip_runtime.h>

#include <iostream>

#include "Exception.h"

#include "init.cuh"

bool init_cuda(void)
{
	size_t size;
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	printf("CUDA heap size found to be: %d bytes\n", (int)size);

	return true;
}