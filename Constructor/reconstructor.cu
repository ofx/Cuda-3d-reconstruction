#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda_types.hpp>
#include <opencv2/cudalegacy.hpp>
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/reduce.hpp"

#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>

#include "VisibleVoxel.h"
#include "cuda_common.cuh"
#include "reconstructor.cuh"

#include "Exception.h"

#define CHECK_ERROR(a) if ((a) != hipSuccess) { goto error; }

#define DIV 2

static VisibleVoxel *sd_visible_voxel_storage;

static unsigned int sh_num_cameras;

static unsigned int sh_width;
static unsigned int sh_height;
static unsigned int sh_depth;

static unsigned int sh_step;

static int sh_x_l;
static int sh_y_l;
static int sh_z_l;

static unsigned int sh_frustum_width;
static unsigned int sh_frustum_height;

float *sd_r = 0, *sd_t = 0, *sd_a = 0, *sd_k = 0;

static bool s_IsInitialized = false;

__device__ short2 project_points_for_camera_kernel(
	float3 point,
	float *R, 
	float *t,
	float *a,
	float *k,
	int camIdx
	)
{
	float fx, fy, cx, cy;

	fx = a[0]; fy = a[4];
	cx = a[2]; cy = a[5];

	float X = point.x, Y = point.y, Z = point.z;
	float x = R[0] * X + R[1] * Y + R[2] * Z + t[0];
	float y = R[3] * X + R[4] * Y + R[5] * Z + t[1];
	float z = R[6] * X + R[7] * Y + R[8] * Z + t[2];
	float r2, r4, r6, a1, a2, a3, cdist, icdist2;
	float xd, yd;

	z = z ? 1.0f / z : 1;
	x *= z; y *= z;

	r2 = x * x + y * y;
	r4 = r2 * r2;
	r6 = r4 * r2;
	a1 = 2 * x*y;
	a2 = r2 + 2 * x * x;
	a3 = r2 + 2 * y * y;
	cdist = 1 + k[0] * r2 + k[1] * r4 + k[4] * r6;

	float k5 = k[5], k6 = k[6], k7 = k[7];
	icdist2 = 1.0f / (1.0f + k5 * r2 + k6 * r4 + k7 * r6);
	xd = x * cdist * icdist2 + k[2] * a1 + k[3] * a2 + k[8] * r2 + k[9] * r4;
	yd = y * cdist * icdist2 + k[2] * a3 + k[3] * a1 + k[10] * r2 + k[11] * r4;

	return make_short2(__float2int_ru(xd * fx + cx), __float2int_ru(yd * fy + cy));
}

__global__
void update_voxels_kernel(
	VisibleVoxel					  *visible_voxel_storage, //
	const cv::cuda::PtrStepSz<uchar>  foregrounds[], 		 // Array of foreground images from cameras
	const cv::cuda::PtrStepSz<uchar3> frames[], 		     // Array of frames from cameras
	float							  *r,
	float							  *t,
	float							  *a,
	float							  *k,
	const unsigned int				  num_cameras,			 // Number of cameras
	const unsigned int				  width,
	const unsigned int                height,
	const unsigned int                depth,
	const int						  x_l,
	const int						  y_l,
	const int						  z_l,
	const unsigned int				  frustum_width,
	const unsigned int				  frustum_height,
	const unsigned int                step,
	unsigned long long int  	      *voxel_pointer,
	const unsigned int				  m_x,
	const unsigned int				  m_y,
	const unsigned int				  m_z,
	const unsigned int				  part
	)
{
	const unsigned int xIdx = (m_x * (width / part)) + blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int yIdx = (m_y * (height / part)) + blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int zIdx = (m_z * (depth / part)) + blockIdx.z * blockDim.z + threadIdx.z;

	const int x = x_l + xIdx * step;
	const int y = y_l + yIdx * step;
	const int z = z_l + zIdx * step;

	int t_r, t_g, t_b;
	t_r = t_g = t_b = 0;

	int v = 0;
	for (int i = 0 ; i < num_cameras ; ++i)
	{
		float3 p;
		p.x = x;
		p.y = y;
		p.z = z;

		float R[9], T[3], A[9], K[12];
		memcpy(R, r + (i * 9), sizeof(float) * 9);
		memcpy(T, t + (i * 3), sizeof(float) * 3);
		memcpy(A, a + (i * 9), sizeof(float) * 9);
		memcpy(K, k + (i * 12), sizeof(float) * 12);

		short2 point = project_points_for_camera_kernel(p, R, T, A, K, i);
		if ((point.x >= 0 && point.x < frustum_width && point.y >= 0 && point.y < frustum_height))
		{
			// Has white pixel in matte?
			uchar pixel = foregrounds[i](point.y, point.x);
			if (pixel == 255)
			{
				++v;

				t_r += frames[i](point.y, point.x).x;
				t_g += frames[i](point.y, point.x).y;
				t_b += frames[i](point.y, point.x).z;
			}
		}
	}

	if (v >= num_cameras)
	{
		unsigned long long int vIdx = atomicAdd(voxel_pointer, 1);

		// Push the voxel into the set of visible voxels
		visible_voxel_storage[vIdx].X = x;
		visible_voxel_storage[vIdx].Y = y;
		visible_voxel_storage[vIdx].Z = z;

		visible_voxel_storage[vIdx].R = t_r / v;
		visible_voxel_storage[vIdx].G = t_g / v;
		visible_voxel_storage[vIdx].B = t_b / v;
	}
}

bool update_voxels(
	const cv::cuda::GpuMat *h_gputmat_foregrounds,
	const cv::cuda::GpuMat *h_gputmat_frames,
	unsigned long long int *h_num_voxels,
	VisibleVoxel		   **h_visible_voxels
	)
{
	cv::cuda::PtrStepSz<uchar> *h_foregrounds = new cv::cuda::PtrStepSz<uchar>[sh_num_cameras];
	cv::cuda::PtrStepSz<uchar3> *h_frames = new cv::cuda::PtrStepSz<uchar3>[sh_num_cameras];
	for (int i = 0 ; i < sh_num_cameras ; ++i)
	{
		h_foregrounds[i] = h_gputmat_foregrounds[i];
		h_frames[i] = h_gputmat_frames[i];
	}

	unsigned long long int h_voxel_pointer, *d_voxel_pointer;
	hipMalloc((void**)&d_voxel_pointer, sizeof(int));
	h_voxel_pointer = 0;
	hipMemcpy(d_voxel_pointer, &h_voxel_pointer, sizeof(unsigned long long int), hipMemcpyHostToDevice);

	cv::cuda::PtrStepSz<uchar> *d_foregrounds = 0;
	CHECK_ERROR(hipMalloc((void**)&d_foregrounds, sizeof(cv::cuda::PtrStepSz<uchar>) * sh_num_cameras));
	CHECK_ERROR(hipMemcpy(d_foregrounds, h_foregrounds, sizeof(cv::cuda::PtrStepSz<uchar>) * sh_num_cameras, hipMemcpyHostToDevice));

	cv::cuda::PtrStepSz<uchar3> *d_frames = 0;
	CHECK_ERROR(hipMalloc((void**)&d_frames, sizeof(cv::cuda::PtrStepSz<uchar3>) * sh_num_cameras));
	CHECK_ERROR(hipMemcpy(d_frames, h_frames, sizeof(cv::cuda::PtrStepSz<uchar3>) * sh_num_cameras, hipMemcpyHostToDevice));

	*h_visible_voxels = NULL;

	// Divide the voxel space into equal divisions to reducs vram usage
	long total_voxels = 0;
	for (int x = 0 ; x < DIV ; ++x)
	{
		for (int y = 0 ; y < DIV ; ++y)
		{
			for (int z = 0 ; z < DIV ; ++z)
			{
				dim3 block_size(16, 8, 8);
				dim3 grid_size = dim3(iDivUp(sh_width / DIV, block_size.x), iDivUp(sh_height / DIV, block_size.y), iDivUp(sh_depth / DIV, block_size.z));
				update_voxels_kernel <<<grid_size, block_size>>>(
					sd_visible_voxel_storage,
					d_foregrounds,
					d_frames,
					sd_r,
					sd_t,
					sd_a,
					sd_k,
					sh_num_cameras,
					sh_width,
					sh_height,
					sh_depth,
					sh_x_l,
					sh_y_l,
					sh_z_l,
					sh_frustum_width,
					sh_frustum_height,
					sh_step,
					d_voxel_pointer,
					x,
					y,
					z,
					DIV
				);

				if (hipDeviceSynchronize() != hipSuccess)
				{
					std::cout << "Failed to initialize voxels..." << std::endl;
					goto error;
				}

				// Fetch number of visible voxels from kernel
				hipMemcpy(&h_voxel_pointer, d_voxel_pointer, sizeof(int), hipMemcpyDeviceToHost);

				// Create memory to store the visible voxels
				*h_visible_voxels = (VisibleVoxel*) realloc(*h_visible_voxels, sizeof(VisibleVoxel) * (h_voxel_pointer + total_voxels));

				//std::cout << "Reallocating " << (sizeof(VisibleVoxel) * (h_voxel_pointer + total_voxels)) / 1000000 << " MB" << std::endl; 

				// Create memory and download visible voxels, store with offset
				hipMemcpy(*h_visible_voxels + total_voxels, sd_visible_voxel_storage, sizeof(VisibleVoxel) * h_voxel_pointer, hipMemcpyDeviceToHost);

				total_voxels += h_voxel_pointer;
			}
		}
	}

	*h_num_voxels = total_voxels;

	// House keeping
	delete[] h_foregrounds;
	delete[] h_frames;

	hipFree(d_frames);
	hipFree(d_foregrounds);
	hipFree(d_voxel_pointer);

	return EXIT_SUCCESS;
error:
	hipError_t err = hipGetLastError();

	char b[500];
	sprintf(b, "Failed to update voxels: %s", hipGetErrorString(err));
	throw_line(b);

	return EXIT_FAILURE;
}

bool initialize_voxels(
	float			       *h_r,
	float                  *h_t,
	float				   *h_a,
	float				   *h_k,
	const unsigned int	   num_cameras,
	const int			   x_l,
	const int			   x_r,
	const int			   y_l,
	const int              y_r, 
	const int              z_l,
	const int              z_r,
	const unsigned int     step,
	const unsigned int     frustum_width,
	const unsigned int     frustum_height,
	int					   *total_voxels
	)
{
	// Compute the dimensions of the voxel space
	const unsigned int width = x_r - x_l;
	const unsigned int height = y_r - y_l;
	const unsigned int depth = z_r - z_l;

	// Compute the storage dimensions of the voxel space
	const unsigned int voxel_space_width = width / step;
	const unsigned int voxel_space_height = height / step;
	const unsigned int voxel_space_depth = depth / step;

	sh_width = voxel_space_width;
	sh_height = voxel_space_height;
	sh_depth = voxel_space_depth;

	sh_frustum_width = frustum_width;
	sh_frustum_height = frustum_height;

	sh_step = step;

	sh_x_l = x_l;
	sh_y_l = y_l;
	sh_z_l = z_l;

	unsigned long long int num_voxels = voxel_space_width;
	num_voxels *= voxel_space_height;
	num_voxels *= voxel_space_depth;
	num_voxels /= pow(DIV, 3);

	*total_voxels = num_voxels;

	std::cout << "Number of voxels per CUDA kernel: " << num_voxels << std::endl;
	std::cout << "Total number of voxels: " << num_voxels * pow(DIV, 3) << std::endl;

	sh_num_cameras = num_cameras;

	// Since when we're destroying we're deallocating memory of voxel storage, we state that we're intitialized at this point
	s_IsInitialized = true;

	// Create storage for visible voxels (device)
	std::cout << "Allocating " << (num_voxels * sizeof(VisibleVoxel)) / 1000000 << " MB of memory for visible voxel storage" << std::endl;
	if (hipMalloc((void**)&sd_visible_voxel_storage, (num_voxels * sizeof(VisibleVoxel))) != hipSuccess)
	{
		goto error;
	}

	// Copy host R, T, K and D to device
	hipMalloc((void**)&sd_r, sizeof(float) * num_cameras * 9);
	hipMalloc((void**)&sd_t, sizeof(float) * num_cameras * 3);
	hipMalloc((void**)&sd_a, sizeof(float) * num_cameras * 9);
	hipMalloc((void**)&sd_k, sizeof(float) * num_cameras * 12);

	// Copy
	hipMemcpy(sd_r, h_r, sizeof(float) * num_cameras * 9, hipMemcpyHostToDevice);
	hipMemcpy(sd_t, h_t, sizeof(float) * num_cameras * 3, hipMemcpyHostToDevice);
	hipMemcpy(sd_a, h_a, sizeof(float) * num_cameras * 9, hipMemcpyHostToDevice);
	hipMemcpy(sd_k, h_k, sizeof(float) * num_cameras * 12, hipMemcpyHostToDevice);

	return EXIT_SUCCESS;
error:
	hipError_t err = hipGetLastError();

	// Clean up
	destroy_voxels();

	char b[500];
	sprintf(b, "Failed to initialize voxels: %s\nVoxel storage is destroyed", hipGetErrorString(err));
	throw_line(b);

	return EXIT_FAILURE;
}

bool destroy_voxels(void)
{
	if (!s_IsInitialized)
	{
		std::cout << "Nothing to destroy!" << std::endl;
		return EXIT_FAILURE;
	}

	// Free the voxel storage
	hipFree(sd_visible_voxel_storage);

	hipFree(sd_a);
	hipFree(sd_k);
	hipFree(sd_t);
	hipFree(sd_r);

	return EXIT_SUCCESS;
}